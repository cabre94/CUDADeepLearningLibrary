#include "hip/hip_runtime.h"
#ifndef ACTIVATION_H
#define ACTIVATION_H

#include <iostream>
#include <string>
#include <stdio.h>
#include "Matrix.h"

__device__ __host__ float sigmoid(int x);

__global__ void sigmoidKernel(float *d_in, float *d_out, int size);

/* ----------------------------
Activation class
---------------------------- */
class Activation{
private:
    std::string name;
    
public:
	Activation(std::string name_);	//Default constructor
	virtual ~Activation();

	std::string getName();
	virtual void call(Matrix &in, Matrix &out) = 0;
};

Activation::Activation(std::string name_) : name(name_) {}

Activation::~Activation(){}

std::string Activation::getName(){
    return name;
}

/* ----------------------------
Sigmoid class
---------------------------- */
class Sigmoid : public Activation{
public:
	Sigmoid();
    ~Sigmoid();

	void call(Matrix &in, Matrix &out);
};

Sigmoid::Sigmoid():Activation("Sigmoid") {}

Sigmoid::~Sigmoid(){}

void Sigmoid::call(Matrix &in, Matrix &out){
	int dev;
	hipGetDevice(&dev);

	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

	// dim3 nThreads(256);
	dim3 nThreads(deviceProp.maxThreadsDim[0]);
	dim3 nBlocks((in.size + nThreads.x - 1) / nThreads.x);
	if(nBlocks.x > deviceProp.maxGridSize[0]){
		nBlocks.x = deviceProp.maxGridSize[0];
	}

	// sigmoidKernel<<< 1, 6 >>>(A.d_elem, A.size);
	sigmoidKernel<<< nBlocks, nThreads >>>(in.getDeviceData(), out.getDeviceData(), in.size);
}



__device__ __host__ float sigmoid(float x){
	return 1.0f / (1 + expf(-x));
}

__global__ void sigmoidKernel(float *d_in, float *d_out, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	while(i < size){
		d_out[i] = sigmoid(d_in[i]);
		i += blockDim.x*gridDim.x;
	}
}




#endif
