#include "hip/hip_runtime.h"
#ifndef LAYERS_H
#define LAYERS_H

#include <iostream>
#include <string>
#include <stdio.h>
#include <stdexcept>
#include "Matrix.cu"
#include "Activation.cu"

/* ----------------------------
Layer class
---------------------------- */
class Layer{
private:
    std::string name;
    
public:
	Layer(std::string name_);	//Default constructor
	virtual ~Layer();
	
	std::string getName();
	// virtual void call(Matrix &in, Matrix &out) = 0;
	// virtual void gradient(Matrix &in, Matrix &out) = 0;
	virtual void printWeights() = 0;
};

Layer::Layer(std::string name_) : name(name_) {}

Layer::~Layer(){}

std::string Layer::getName(){return name;}


/* ----------------------------
Dense Layer
---------------------------- */

class Dense : public Layer{
private:
	Matrix W;
	Matrix b;
	Activation *activation;
public:
	Dense(int width, int height, std::string act, std::string dist = "uniform", float w = 0.1);
    ~Dense();
	
	// void call(Matrix &in, Matrix &out);
	// void gradient(Matrix &in, Matrix &out);
	void printWeights();
};

Dense::Dense(int width, int height, std::string act, std::string dist, float w)
	:Layer("Dense"), W(width,height,dist,w), b(1,height,"ones",0) {
		if(act == "linear")
			activation = new Linear;
		else if(act == "relu")
			activation = new Relu;
		else if(act == "sigmoid")
			activation = new Sigmoid;
		else if(act == "tanh")
			activation = new Tanh;
		else if(act == "leakyRelu")
			activation = new LeakyRelu();
		else
			throw std::invalid_argument("Invalid activation");
	}

Dense::~Dense(){
	delete activation;
}

void Dense::printWeights(){
	float *ptr_W = W.getHostData();
	float *ptr_b = b.getHostData();
	for(int i=0; i < W.height; ++i){
		for(int j=0; j < W.width; ++j)
			std::cout << ptr_W[i*W.width + j] << "\t";
		std::cout << ptr_b[i] << "\t";
		std::cout << std::endl;
	}
}


/* ----------------------------
Input Layer
---------------------------- */

class Input : public Layer{
private:
	int out_width, out_height;
public:
	Input(int width, int height);
    ~Input();
	
	// void call(Matrix &in, Matrix &out);
	// void gradient(Matrix &in, Matrix &out);
	void printWeights();
};

Input::Input(int width, int height):Layer("Input"), out_width(width), out_height(height){}

Input::~Input(){}

void Input::printWeights(){
	std::cout << "Input Layer?" << std::endl;
}






#endif
