#ifndef MATRIX_H
#define MATRIX_H

#include <iostream>
#include <string>
#include <stdio.h>
#include <random>


class Matrix{
public:
	int height, width, size;
	float *h_elem, *d_elem;
	float weight;
	std::string dist;

	bool allocated;

// public:
	Matrix();
	Matrix(int height, int width, std::string dist = "uniform", float w = 1);
	~Matrix();

	void copyDeviceToHost();
	void copyHostToDevice();
	void print();

	float* getHostData();
	float* getDeviceData();

	int getHeight();
	int getWidth();

	void initialize(int height, int width, std::string dist = "uniform", float w = 1);
};

Matrix::Matrix(){
	allocated = false;
}

Matrix::Matrix(int height, int width, std::string dist, float w)
			: height(height), width(width), size(width * height){
	weight = w;
	dist = dist;
	h_elem = new float[size];
	
	std::random_device rd;
	std::mt19937 mt(rd());

	if(dist == "normal"){
		// std::default_random_engine generator;
  		std::normal_distribution<float> distribution(0.0,weight);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "uniform"){
		// std::default_random_engine generator;
		std::uniform_real_distribution<float> distribution(-weight,1.0);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "ones"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 1.0f;
		}
	}else if(dist == "zeros"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 0.0f;		}
	}else{
		throw std::invalid_argument("Invalid Weight initialization");
	}

	// Allocacion en device
	hipMalloc(&d_elem, size * sizeof(float));
	hipMemcpy( d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice);

	allocated = true;
}

Matrix::~Matrix(){
	delete [] h_elem;
	hipFree(d_elem);
}

void Matrix::copyDeviceToHost(){
	hipMemcpy(h_elem, d_elem, size * sizeof(float), hipMemcpyDeviceToHost);
}

void Matrix::copyHostToDevice(){
	hipMemcpy(d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice );
}

void Matrix::print(){
	for(int i=0; i < height; ++i){
		for(int j=0; j < width; ++j)
			std::cout << h_elem[i*width + j] << "\t";
		std::cout << std::endl;
	}
}

float* Matrix::getHostData(){
	return h_elem;
}

float* Matrix::getDeviceData(){
	return d_elem;
}

int Matrix::getHeight(){return height;}

int Matrix::getWidth(){return width;}

void Matrix::initialize(int height_, int width_, std::string dist, float w){
	if (allocated){
		delete [] h_elem;
		hipFree(d_elem);
		allocated = false;
	}

	height = height_;
	width = width_;
	size = width * height;
	weight = w;
	dist = dist;
	h_elem = new float[size];
	
	std::random_device rd;
	std::mt19937 mt(rd());

	if(dist == "normal"){
		// std::default_random_engine generator;
  		std::normal_distribution<float> distribution(0.0,weight);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "uniform"){
		// std::default_random_engine generator;
		std::uniform_real_distribution<float> distribution(-weight,1.0);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "ones"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 1.0f;
		}
	}else if(dist == "zeros"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 0.0f;		}
	}else{
		throw std::invalid_argument("Invalid Weight initialization");
	}

	// Allocacion en device
	hipMalloc(&d_elem, size * sizeof(float));
	hipMemcpy( d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice);

	allocated = true;
}



#endif
