#ifndef MATRIX_H
#define MATRIX_H

#include <iostream>
#include <string>
#include <stdio.h>
#include <random>

// Copia los elementos de to a from
__global__ void copyFromTo(float *from, float *to, int size);
// Copia los elementos de las filas idx de to a from
__global__ void copyFromToIdx(float *from, float *to, int *idx, int h, int w, int idx0);

class Matrix{
public:
	int height, width, size;
	float *h_elem, *d_elem;
	float weight;
	std::string dist;

	bool allocated;

// public:
	Matrix();
	Matrix(int height, int width, std::string dist = "uniform", float w = 1);
	~Matrix();

	void copyDeviceToHost();
	void copyHostToDevice();
	void print();
	void printDimensions();

	float* getHostData();
	float* getDeviceData();

	int getHeight();
	int getWidth();

	void initialize(int height, int width, std::string dist = "zeros", float w = 1);
	void copyDeviceDataFromAnother(Matrix &from);
	void copyDeviceDataFromBatch(Matrix &from, int *idx, int idx0);
};

Matrix::Matrix(){
	allocated = false;
}

Matrix::Matrix(int height, int width, std::string dist, float w)
			: height(height), width(width), size(width * height){
	weight = w;
	dist = dist;
	h_elem = new float[size];
	
	std::random_device rd;
	std::mt19937 mt(rd());

	if(dist == "normal"){
		// std::default_random_engine generator;
  		std::normal_distribution<float> distribution(0.0,weight);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "uniform"){
		// std::default_random_engine generator;
		std::uniform_real_distribution<float> distribution(-weight,1.0);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "ones"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 1.0f;
		}
	}else if(dist == "zeros"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 0.0f;		}
	}else{
		throw std::invalid_argument("Invalid Weight initialization");
	}

	// Allocacion en device
	hipMalloc(&d_elem, size * sizeof(float));
	hipMemcpy( d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice);

	allocated = true;
}

Matrix::~Matrix(){
	if(allocated){
		delete [] h_elem;
		hipFree(d_elem);
	}
}

void Matrix::copyDeviceToHost(){
	hipMemcpy(h_elem, d_elem, size * sizeof(float), hipMemcpyDeviceToHost);
}

void Matrix::copyHostToDevice(){
	hipMemcpy(d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice );
}

void Matrix::print(){
	for(int i=0; i < height; ++i){
		for(int j=0; j < width; ++j)
			std::cout << h_elem[i*width + j] << "\t";
		std::cout << std::endl;
	}
}

void Matrix::printDimensions(){
	std::cout << "(" << height << "," << width << ")";
}

float* Matrix::getHostData(){
	return h_elem;
}

float* Matrix::getDeviceData(){
	return d_elem;
}

int Matrix::getHeight(){return height;}

int Matrix::getWidth(){return width;}

void Matrix::initialize(int height_, int width_, std::string dist, float w){
	if (allocated){
		delete [] h_elem;
		hipFree(d_elem);
		allocated = false;
	}

	height = height_;
	width = width_;
	size = width * height;
	weight = w;
	dist = dist;
	h_elem = new float[size];
	
	std::random_device rd;
	std::mt19937 mt(rd());

	if(dist == "normal"){
		// std::default_random_engine generator;
  		std::normal_distribution<float> distribution(0.0,weight);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "uniform"){
		// std::default_random_engine generator;
		std::uniform_real_distribution<float> distribution(-weight,1.0);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(mt);
		}
	}else if(dist == "ones"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 1.0f;
		}
	}else if(dist == "zeros"){
		for(int i=0; i < size; ++i){
			h_elem[i] = 0.0f;		}
	}else{
		throw std::invalid_argument("Invalid Weight initialization");
	}

	// Allocacion en device
	hipMalloc(&d_elem, size * sizeof(float));
	hipMemcpy( d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice);

	allocated = true;
}

void Matrix::copyDeviceDataFromAnother(Matrix &from){
	// Asumo dimensiones correctas
	int dev;
	hipGetDevice(&dev);
	
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
	
	// dim3 nThreads(256);
	dim3 nThreads(deviceProp.maxThreadsDim[0]);
	dim3 nBlocks((from.size + nThreads.x - 1) / nThreads.x);
	if(nBlocks.x > deviceProp.maxGridSize[0]){
		nBlocks.x = deviceProp.maxGridSize[0];
	}
	
	copyFromTo<<< nBlocks, nThreads >>>(from.getDeviceData(), d_elem, from.size);
	hipDeviceSynchronize();
	// Aca Host y Device son distintos
}

void Matrix::copyDeviceDataFromBatch(Matrix &from, int *idx, int idx0){
	// Asumo dimensiones correctas
	int dev;
	hipGetDevice(&dev);
	
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

	// dim3 nThreads(256);
	// dim3 nBlocks((N + nThreads.x - 1) / nThreads.x);

	// dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	
	// dim3 nThreads(256);
	// dim3 nThreads(deviceProp.maxThreadsDim[0]);
	// dim3 nBlocks((from.size + nThreads.x - 1) / nThreads.x);
	// if(nBlocks.x > deviceProp.maxGridSize[0]){
	// 	nBlocks.x = deviceProp.maxGridSize[0];
	// }
	dim3 dimBlock(deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1]);
	dim3 dimGrid(width/dimBlock.x, height/dimBlock.y);
	if(dimGrid.x > deviceProp.maxGridSize[0]){
		dimGrid.x = deviceProp.maxGridSize[0];
	}
	if(dimGrid.y > deviceProp.maxGridSize[1]){
		dimGrid.y = deviceProp.maxGridSize[1];
	}
	
	copyFromToIdx<<< dimGrid, dimBlock >>>(from.getDeviceData(), d_elem, idx, height, width, idx0);
	hipDeviceSynchronize();
}




/* ----------------------------
Kernels
---------------------------- */


__global__ void copyFromTo(float *from, float *to, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	while(i < size){
		to[i] = from[i];

		i += blockDim.x * gridDim.x;
	}
}

// from es mas grande
// las filas de to son iguales a el tamaño
// idx deberia tener h elementos

__global__ void copyFromToIdx(float *from, float *to, int *idx, int h, int w, int idx0){
	// lo tengo que hacer como columna
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row < h && col < w){
		to[row * w + col] = from[idx[idx0+row] * w + col];
	}
}



#endif
