#ifndef MATRIX_H
#define MATRIX_H

#include <iostream>
#include <string>
#include <stdio.h>
#include <random>


class Matrix{
public:
	int height, width, size;
	float *h_elem, *d_elem;
	float weight;
	std::string dist;

// public:
	Matrix(int width, int height, std::string dist_ = "uniform", float w = 1);
	~Matrix();

	void copyDeviceToHost();
	void copyHostToDevice();
	void print();

	float* getDeviceData();
};


Matrix::Matrix(int height, int width, std::string dist_, float w)
			: height(height), width(width), size(width * height){
	weight = w;
	dist = dist_;
	h_elem = new float[size];
	
	// float aux[3] = {-2, 0 , 2};
	// for(int i=0; i < size; ++i){
	// 	// h_elem[i] = i;
	// 	// h_elem[i] = 0;
	// 	h_elem[i] = aux[i%3];
	// }
	if(dist == "uniform"){
		std::default_random_engine generator;
  		std::normal_distribution<float> distribution(0.0,weight);
		for(int i=0; i < size; ++i){
			h_elem[i] = distribution(generator);
		}
	}

	// Allocacion en device
	hipMalloc(&d_elem, size * sizeof(float));
	hipMemcpy( d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice);
}

Matrix::~Matrix(){
	delete [] h_elem;
	hipFree(d_elem);
}

void Matrix::copyDeviceToHost(){
	hipMemcpy(h_elem, d_elem, size * sizeof(float), hipMemcpyDeviceToHost);
}

void Matrix::copyHostToDevice(){
	hipMemcpy(d_elem, h_elem, size * sizeof(float), hipMemcpyHostToDevice );
}

void Matrix::print(){
	for(int i=0; i < height; ++i){
		for(int j=0; j < width; ++j)
			std::cout << h_elem[i*width + j] << "\t";
		std::cout << std::endl;
	}
}

float* Matrix::getDeviceData(){
	return d_elem;
}

#endif
